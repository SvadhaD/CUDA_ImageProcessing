#include "utils.cuh"

void* cudaMallocSafe(size_t size) {
    void* devPtr;
    checkCudaError(hipMalloc(&devPtr, size), "hipMalloc failed");
    return devPtr;
}

void cudaMemcpySafe(void* dst, const void* src, size_t size, hipMemcpyKind kind) {
    checkCudaError(hipMemcpy(dst, src, size, kind), "hipMemcpy failed");
}

void cudaFreeSafe(void* devPtr) {
    checkCudaError(hipFree(devPtr), "hipFree failed");
}
