#include "convolution.cuh"
#include "morphology.cuh"
#include "utils.cuh"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace cv;
using namespace std;

void comparePerformance(const Mat& inputImage, int kernelSize) {
    Mat cpuOutput(inputImage.size(), CV_8UC1);
    Mat gpuOutput(inputImage.size(), CV_8UC1);
    vector<float> kernel(kernelSize * kernelSize, 1.0f / (kernelSize * kernelSize));
    
    // CPU Convolution
    auto start = chrono::high_resolution_clock::now();
    filter2D(inputImage, cpuOutput, -1, Mat(kernelSize, kernelSize, CV_32F, kernel.data()));
    auto end = chrono::high_resolution_clock::now();
    cout << "CPU Convolution Time: " << chrono::duration<double, milli>(end - start).count() << " ms" << endl;
    
    // GPU Convolution
    start = chrono::high_resolution_clock::now();
    applyConvolution(inputImage.data, gpuOutput.data, kernel.data(), inputImage.cols, inputImage.rows, kernelSize);
    end = chrono::high_resolution_clock::now();
    cout << "GPU Convolution Time: " << chrono::duration<double, milli>(end - start).count() << " ms" << endl;
    
    imwrite("cpu_convolution.png", cpuOutput);
    imwrite("gpu_convolution.png", gpuOutput);
}

int main(int argc, char** argv) {
    if (argc != 2) {
        cerr << "Usage: " << argv[0] << " <image_path>" << endl;
        return -1;
    }
    
    Mat inputImage = imread(argv[1], IMREAD_GRAYSCALE);
    if (inputImage.empty()) {
        cerr << "Error: Could not open image!" << endl;
        return -1;
    }
    
    Mat dilatedImage(inputImage.size(), CV_8UC1);
    Mat erodedImage(inputImage.size(), CV_8UC1);
    
    int structuringElement[9] = { 0, 1, 0, 1, 1, 1, 0, 1, 0 };
    
    applyDilation(inputImage.data, dilatedImage.data, structuringElement, inputImage.cols, inputImage.rows, 3);
    applyErosion(inputImage.data, erodedImage.data, structuringElement, inputImage.cols, inputImage.rows, 3);
    
    imwrite("dilated.png", dilatedImage);
    imwrite("eroded.png", erodedImage);
    
    comparePerformance(inputImage, 3);
    
    cout << "Processing complete. Output images saved." << endl;
    return 0;
}
